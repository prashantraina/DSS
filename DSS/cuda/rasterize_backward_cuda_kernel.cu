#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "macros.hpp"
#include <ATen/ExpandUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/core/ScalarType.h>
#include <stdio.h>
#include <torch/extension.h>

/*
 return the indice of current point in the idxList
 -1 outside
 >= 0 inside
 */
template <typename indice_t>
__device__ void is_inside(const int topK, const indice_t *__restrict__ idxList,
                          const indice_t curr_Idx, int *curK) {
  for (size_t i = 0; i < topK; i++) {
    // a pixel is inside the splat if idxList contains point index
    if (idxList[i] == curr_Idx) {
      *curK = i;
      return;
    }
    // a pixel definitely isn't inside a splat if it's not occupied by any point
    if (idxList[i] == -1) {
      *curK = -1;
      return;
    }
  }
  *curK = -1;
  return;
}

/* compute pixel color after removing a point from a merged pixel */
// TODO curPointList probably no necessary, since rhoList and WsList will be
// zero at curPointList[k] == -1
template <typename scalar_t, typename indice_t>
__device__ void after_removal(const int numColors, const int topK,
                              const int curK, const scalar_t depthThres,
                              const scalar_t *depthList,
                              const indice_t *curPointList, // topK
                              const uint8_t *curIsBehind,   // topK
                              const scalar_t *wsList,       // topKx3
                              const scalar_t *rhoList,      // topKx1
                              const scalar_t *curPixel,     // numColors
                              scalar_t *newColors,          // numColors
                              scalar_t *newDepth) {
  // initialize color with 0.0
  for (int c = 0; c < numColors; c++) 
    newColors[c] = 0.0;
  
  // initialize depth with the farthest so far
  *newDepth = depthList[topK - 1];

  scalar_t sumRho = 0.0;
  int numVisible = 0;
  for (int k = 0; k < topK; k++) {
    if (curIsBehind[k] == 0)
      ++numVisible;
  }
  // if it's the only visible point, then removing it will reveal the
  // color below
  assert(numVisible >= 0);
  if (numVisible == 1) 
  {
    sumRho = 0.0;
    // CHECK: should be the second?
    const scalar_t curDepth = depthList[1];
    
    for(int k = curK + 1; k < topK; k++) 
    {
      // as soon as idxList is -1 or depth > currentDepth+threshold
      // stop accumulating colors
      if (curPointList[k] == -1) 
        break;

      if ((depthList[k] - curDepth) > depthThres) 
        break;
        
      for (int c = 0; c < numColors; c++) 
        newColors[c] += wsList[k * numColors + c] * rhoList[k];
      
      sumRho += rhoList[k];

      if (depthList[k] < *newDepth) 
        *newDepth = depthList[k];
    }

    for (int c = 0; c < numColors; c++) 
      newColors[c] /= (sumRho + 1e-8);

    return;
  }

  // not the only point visible:
  // removing current point involves reweighting rhos
  for (size_t k = 0; k < numVisible; k++) 
  {
    if (k == curK) 
      continue;
      
    for (size_t c = 0; c < numColors; c++) 
      newColors[c] += wsList[k * numColors + c] * rhoList[k];
      
    sumRho += rhoList[k];

    if (depthList[k] < *newDepth) 
      *newDepth = depthList[k];
  }

  for (size_t c = 0; c < numColors; c++) 
    newColors[c] /= (sumRho + 1e-8);
    
  assert(sumRho > 0);
  return;
}

/* compute pixel color after moving a point to a merged pixel */
template <typename scalar_t>
__device__ void
after_addition(const int numColors, const int topK, const scalar_t rho,
               const scalar_t *ws, const scalar_t pointDepth,
               const scalar_t depthThres, const scalar_t *depthList,
               const uint8_t *curIsBehind, // topK
               const scalar_t *wsList,     // topKx3
               const scalar_t *rhoList,    // topKx1
               const scalar_t *curPixel,   // numColors
               scalar_t *newColors,        // numColors
               scalar_t *newDepth) {
  scalar_t sumRho = rho;
  for (size_t k = 0; k < topK; k++) {
    if (curIsBehind[k] > 0 ||
        (depthList[k] - depthThres) > pointDepth) { // || (depthList[k] - depthThres) > pointDepth
      break;
    }
    sumRho += rhoList[k];
  }

  if (sumRho == 0) {
    sumRho += 1e-5;
  }

  for (size_t c = 0; c < numColors; c++) 
    newColors[c] = rho / sumRho * ws[c];

  for (size_t k = 0; k < topK; k++) {
    for (size_t c = 0; c < numColors; c++) {
      if (curIsBehind[k] > 0 ||
          (depthList[k] - depthThres) >
              pointDepth) { // || (depthList[k] - depthThres) > pointDepth
        break;
      }
      newColors[c] += rhoList[k] / sumRho * wsList[k * numColors + c];
    }
  }
  *newDepth = min(depthList[0], pointDepth);
}

/*
  compute pixel color after moving a point closer to the screen
 */
template <typename scalar_t>
__device__ void after_drawing_closer(const int numColors, const int topK,
                                     const int curK,
                                     const scalar_t *wsList,    // topKx3
                                     const scalar_t *rhoList,   // topKx1
                                     const scalar_t *depthList, // topK
                                     const uint8_t *isBehind,   // topK
                                     scalar_t *newColors, scalar_t *newDepth) {
  scalar_t curRho = rhoList[curK];
  const scalar_t *curW = wsList + curK * numColors;
  scalar_t pointDepth = depthList[curK];
  scalar_t sumRho = curRho;
  for (size_t k = 0; k < topK; k++) {
    if (isBehind[k] > 0) {
      break;
    }
    sumRho += rhoList[k];
  }
  // should at least have curRho
  assert(sumRho > 0);
  for (size_t c = 0; c < numColors; c++) {
    newColors[c] = curRho / sumRho * curW[c];
  }

  for (size_t k = 0; k < topK; k++) {
    for (size_t c = 0; c < numColors; c++) {
      if (isBehind[k] > 0) {
        break;
      }
      newColors[c] += rhoList[k] / sumRho * wsList[k * numColors + c];
    }
  }
  *newDepth = min(depthList[0], pointDepth);
}

template <typename scalar_t> __device__ scalar_t eps_guard(scalar_t v) {
  const scalar_t eps = 0.01;
  if (v < 0) {
    return v - eps;
  }
  if (v >= 0) {
    return v + eps;
  }
  // return v;
}
/*
  a point is not "bad", i.e. don't need to be moved, when it's colorGrads is
  zero within its effective extent (pointIdxMap include pointID && rhoMap > 0)
 */
template <typename scalar_t, typename indice_t>
__global__ void whitelist_points(
    const int imgHeight, const int imgWidth, const int topK, const int PN,
    const int batchSize, const int WDim,
    const scalar_t *__restrict__ colorGrads,    // BxHxWx3 gradient from output
    const indice_t *__restrict__ pointIdxMap,   // BxHxWxtopK
    const uint8_t *__restrict__ isBehind,       // BxHxWxtopK
    const indice_t *__restrict__ boundingBoxes, // BxNx4 xmin ymin xmax ymax
    uint8_t *whitelist_mask                     // BxNx1
) {
  const int numPixels = imgHeight * imgWidth;
  const scalar_t eps = 1e-9;
  // loop all points
  for (int b = blockIdx.x; b < batchSize; b += gridDim.x) {
    for (indice_t p = threadIdx.x + blockDim.x * blockIdx.y; p < PN;
         p += blockDim.x * gridDim.y) {
      const size_t curPointIdx = b * PN + p;
      const indice_t *curBB = boundingBoxes + curPointIdx * 4;
      const indice_t xmin = curBB[0];
      const indice_t ymin = curBB[1];
      const indice_t xmax = curBB[2];
      const indice_t ymax = curBB[3];
      // search within the bounding box
      bool isGood = true;
      bool inExtent = false;
      for (size_t h = ymin; h < ymax; h++) {
        for (size_t w = xmin; w < xmax; w++) {
          const indice_t curPixelIdx = b * numPixels + h * imgWidth + w;
          scalar_t colorGrad = 0.0;
          for (size_t c = 0; c < WDim; c++) {
            colorGrad += abs(colorGrads[curPixelIdx * WDim + c]);
          }
          // temporary flag for current pixel
          bool _isGood = true;
          bool _inExtent = false;
          for (size_t k = 0; k < topK; k++) {
            // inside the extent and is shown
            if (pointIdxMap[curPixelIdx * topK + k] == p) {
              _inExtent = true;
              // is bad if some pixel inside the splat radius is not shown
              // (isBehind) or colorGrad > threshold
              _isGood =
                  !(isBehind[curPixelIdx * topK + k] > 0 || (colorGrad) > eps);
            }
          }
          // there is one pixel in extent
          inExtent = inExtent | _inExtent;
          // as long as one pixel is not good, this point is not good
          isGood = _isGood & isGood;
        }
      }
      // if all pixels are not in extent, then this point is bad
      whitelist_mask[curPointIdx] = inExtent & isGood;
    }
  }
}
/*  */
template <typename scalar_t, typename indice_t>
__global__ void visibility_backward_kernel(
    const int batchSize, const int imgHeight, const int imgWidth,
    const int localHeight, const int localWidth, const int topK, const int PN,
    const int projDim, const int WDim, const scalar_t focalL,
    const scalar_t mergeT, const bool considerZ,
    const scalar_t *__restrict__ colorGrads,    // BxHxWxC gradient from output
    const indice_t *__restrict__ pointIdxMap,   // BxHxWxtopK
    const scalar_t *__restrict__ rhoMap,        // BxHxWxtopK
    const scalar_t *__restrict__ wsMap,         // BxHxWxtopKxC
    const scalar_t *__restrict__ depthMap,      // BxHxWxtopK
    const uint8_t *__restrict__ isBehind,       // BxHxWxtopK
    const scalar_t *__restrict__ pixelValues,   // BxHxWxC
    const indice_t *__restrict__ boundingBoxes, // BxNx4 xmin ymin xmax ymax
    const scalar_t *__restrict__ projPoints,    // BxNx[2or3], xy1
    const scalar_t *__restrict__ pointColors,   // BxNxC
    const scalar_t *__restrict__ depthValues,   // BxNx1
    const scalar_t *__restrict__ rhoValues,     // BxNx1
    scalar_t *__restrict__ dIdp, // BxNx2 gradients for screenX and screenY
    scalar_t *__restrict__ dIdz) // BxNx1 gradients for z
{
  // const scalar_t mergeT = scalar_t(mergeThres);
  // const scalar_t focalL = scalar_t(focalLength);
  const int numPixels = imgHeight * imgWidth;
  // loop all points
  for (int b = blockIdx.x; b < batchSize; b += gridDim.x) {
    for (indice_t p = threadIdx.x + blockDim.x * blockIdx.y; p < PN;
         p += blockDim.x * gridDim.y) {
      const indice_t curPointIdx = b * PN + p;
      // skip point (gradient=0) if mask == 1 (i.e. point is good)
      scalar_t xmin = scalar_t(boundingBoxes[curPointIdx * 4]);
      scalar_t ymin = scalar_t(boundingBoxes[curPointIdx * 4 + 1]);
      // scalar_t xmax = scalar_t(boundingBoxes[curPointIdx * 4 + 2]);
      // scalar_t ymax = scalar_t(boundingBoxes[curPointIdx * 4 + 3]);
      const scalar_t *curPointColor = pointColors + curPointIdx * WDim;
      const scalar_t *curProjValues = projPoints + curPointIdx * projDim;
      scalar_t *dIdx = dIdp + curPointIdx * projDim;
      scalar_t *dIdy = dIdp + curPointIdx * projDim + 1;
      scalar_t *curdIdz = dIdz + curPointIdx;
      const scalar_t rhov = rhoValues[curPointIdx];
      const int bH =
          min(max(0, int(curProjValues[1] - localHeight / 2)), imgHeight);
      const int eH =
          max(min(imgHeight, int(curProjValues[1] + localHeight / 2 + 1)), 0);
      const int bW =
          min(max(0, int(curProjValues[0] - localWidth / 2)), imgWidth);
      const int eW =
          max(min(imgWidth, int(curProjValues[0] + localWidth / 2 + 1)), 0);
      // loop all pixels
      for (size_t i = bH; i < eH; i++) {
        for (size_t j = bW; j < eW; j++) {
          const indice_t curPixelIdx = (b * numPixels + i * imgWidth + j);
          const scalar_t *curColorGrad = colorGrads + curPixelIdx * WDim;
          const scalar_t *curWs = wsMap + curPixelIdx * topK * WDim;
          const scalar_t *curRhos = rhoMap + curPixelIdx * topK;
          // const indice_t curClosest = pointIdxMap[curPixelIdx * topK];
          // const indice_t curClosestIdx = b * PN + curClosest;
          const indice_t *curIdxList = pointIdxMap + curPixelIdx * topK;
          const scalar_t *curPixelValues = pixelValues + curPixelIdx * WDim;
          const scalar_t *curDepthList = depthMap + curPixelIdx * topK;
          // const scalar_t curClosestDepth = depthMap[curPixelIdx * topK];
          const uint8_t *curIsBehind = isBehind + curPixelIdx * topK;
          const scalar_t curPointDepth = depthValues[curPointIdx];
          // is this pixel inside the splat?
          int curK;
          is_inside(topK, curIdxList, curPointIdx, &curK);
          scalar_t didxv = 0.0;
          scalar_t didyv = 0.0;
          scalar_t didzv = 0.0;
          scalar_t dldI = 0.0;
          scalar_t newColors[10];
          scalar_t newDepth;

          // outside
          if (curK < 0) 
          {
            after_addition(WDim, topK, rhov, curPointColor, curPointDepth,
                           mergeT, curDepthList, curIsBehind, curWs, curRhos,
                           curPixelValues, newColors, &newDepth);

            for (size_t c = 0; c < WDim; c++) 
              dldI += (newColors[c] - curPixelValues[c]) * curColorGrad[c];
              
            if (dldI < 0.0) 
            {
              // another point at pixel i,j is in front of the current point by
              // a threshold, need to change z, otherwise moving to that
              // direction won't change the color value
              if (curPointDepth - newDepth > mergeT) 
              {
                if (!considerZ) 
                  continue;
                  
                scalar_t dx = (scalar_t(j) - curProjValues[0]);
                scalar_t dy = (scalar_t(i) - curProjValues[1]);
                scalar_t dx_3d = (scalar_t(j) - curProjValues[0]) / focalL /
                                 imgWidth * 2 * curPointDepth;
                scalar_t dy_3d = (scalar_t(i) - curProjValues[1]) / focalL /
                                 imgHeight * 2 * curPointDepth;
                assert(newDepth < curPointDepth);
                scalar_t dz_3d = newDepth - curPointDepth;
                scalar_t distance2_3d =
                    eps_guard(dx_3d * dx_3d + dy_3d * dy_3d + dz_3d * dz_3d);
                scalar_t distance2 = eps_guard(dx * dx + dy * dy);
                didzv = dldI / distance2_3d * dz_3d;
                // should rescale to screen space
                didxv = dldI / distance2 * dx;
                didyv = dldI / distance2 * dy;
                assert(!isnan(didxv));
                assert(!isnan(didyv));
              } 
              else // don't need to change z
              {
                scalar_t dx = (scalar_t(j) - curProjValues[0]);
                scalar_t dy = (scalar_t(i) - curProjValues[1]);
                scalar_t distance2 = eps_guard(dx * dx + dy * dy);
                // dIdx
                didxv = dldI / distance2 * dx;
                // dIdy
                didyv = dldI / distance2 * dy;
                assert(!isnan(didxv));
                assert(!isnan(didyv));
              }
            }
          }
          // pixel inside splat
          else // i.e. curK >= 0
          {
            // is the current point shown?
            if (curIsBehind[curK] < 1) // yes
            {
              // dIdx dIdy and dIdz-
              after_removal(WDim, topK, curK, mergeT, curDepthList, curIdxList,
                            curIsBehind, curWs, curRhos, curPixelValues,
                            newColors, &newDepth);

              for (size_t c = 0; c < WDim; c++) 
                dldI += (newColors[c] - curPixelValues[c]) * curColorGrad[c];

              if (dldI < 0.0) 
              {
                // dIdp = (dIdp+) + (dIdp-)
                scalar_t dx = (scalar_t(j) - curProjValues[0]);
                scalar_t dy = (scalar_t(i) - curProjValues[1]);
                scalar_t distance = sqrt(eps_guard(dx * dx + dy * dy));
                scalar_t rx = curProjValues[0] - xmin;
                scalar_t ry = curProjValues[1] - ymin;
                assert(rx > 0);
                assert(ry > 0);
                scalar_t r = max(rx, ry);
                didxv = dldI * dx / eps_guard((r + distance) * distance) +
                        dldI * dx / eps_guard((distance - r) * distance);
                didyv = dldI * dy / eps_guard((r + distance) * distance) +
                        dldI * dy / eps_guard((distance - r) * distance);
                assert(!isnan(didxv));
                assert(!isnan(didyv));
              }
            } // endif (curRhos[curK] > 0)
            // point is not visible:
            else // i.e. curIsBehind[curK] >= 1
            {
              if (!considerZ)
                continue;
              // this point is occluded by other points, moving closer will
              // change the color
              after_drawing_closer(WDim, topK, curK, curWs, curRhos,
                                   curDepthList, curIsBehind, newColors,
                                   &newDepth);
              for (size_t c = 0; c < WDim; c++) {
                dldI += (newColors[c] - curPixelValues[c]) * curColorGrad[c];
              }
              if (dldI < 0.0) {
                didzv = dldI / eps_guard(newDepth - curPointDepth);
              }
            } // endif on top
          }   // endif inside

          (*curdIdz) += didzv;
          (*dIdx) += didxv;
          (*dIdy) += didyv;
        } // imWidth
      }   // imHeight
    }     // point
  }       // batch
}

// dIdp BxNx2 dx dy, dIdz BxNx1
std::vector<at::Tensor>
visibility_backward_cuda(const double focalLength, const double mergeThres,
                         const bool considerZ, const int localHeight,
                         const int localWidth,
                         const at::Tensor &colorGrads,    // BxHxWxWDim
                         const at::Tensor &pointIdxMap,   // BxHxWxtopK
                         const at::Tensor &rhoMap,        // BxHxWxtopK
                         const at::Tensor &wsMap,         // BxHxWxtopKxWDim
                         const at::Tensor &depthMap,      // BxHxWxtopK
                         const at::Tensor &isBehind,      // BxHxWxtopK
                         const at::Tensor &pixelValues,   // BxHxWxWDim
                         const at::Tensor &boundingBoxes, // BxNx4
                         const at::Tensor &projPoints,    // BxNx[2or3]
                         const at::Tensor &pointColors,   // BxNxWDim
                         const at::Tensor &depthValues,   // BxNx1
                         const at::Tensor &rhoValues,     // BxNx1
                         at::Tensor &dIdp, at::Tensor &dIdz) {
  const int batchSize = pointIdxMap.size(0);
  const int imgHeight = pointIdxMap.size(1);
  const int imgWidth = pointIdxMap.size(2);
  const int topK = pointIdxMap.size(3);
  const int PN = projPoints.size(1);
  const int WDim = pointColors.size(2);
  CHECK(projPoints.size(2) == 2 || projPoints.size(2) == 3);
  const int projDim = projPoints.size(2);
  CHECK_EQ(pointColors.size(1), PN);
  CHECK(colorGrads.size(-1) == wsMap.size(-1) &&
        wsMap.size(-1) == pixelValues.size(-1) &&
        pixelValues.size(-1) == pointColors.size(-1));
  std::vector<at::Tensor> outputs;
  unsigned int n_threads, n_blocks;
  n_threads = opt_n_threads(PN);
  n_blocks = min(32, (PN * batchSize + n_threads - 1) / n_threads);
  // initialize with zeros
  dIdp.zero_();
  dIdz.zero_();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  std::vector<at::Tensor> output;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      colorGrads.type(), "visibility_backward_kernel", ([&] {
        visibility_backward_kernel<scalar_t, int64_t>
            <<<dim3(batchSize, n_blocks, 1), n_threads, 0, stream>>>(
                batchSize, imgHeight, imgWidth, localHeight, localWidth, topK,
                PN, projDim, WDim, focalLength, mergeThres, considerZ,
                colorGrads.data<scalar_t>(),  // BxHxWx3
                pointIdxMap.data<int64_t>(),  // BxHxWxtopK
                rhoMap.data<scalar_t>(),      // BxHxWxtopK
                wsMap.data<scalar_t>(),       // BxHxWxtopKx3
                depthMap.data<scalar_t>(),    // BxHxWxtopK
                isBehind.data<uint8_t>(),     // BxHxWxtopK
                pixelValues.data<scalar_t>(), // BxHxWx3
                boundingBoxes.toType(pointIdxMap.scalar_type())
                    .data<int64_t>(),         // BxNx4 xmin ymin xmax ymax
                projPoints.data<scalar_t>(),  // BxNx[2or3], xy1
                pointColors.data<scalar_t>(), // BxNx3
                depthValues.data<scalar_t>(), // BxNx1
                rhoValues.data<scalar_t>(),   // BxNx1
                dIdp.data<scalar_t>(),        // BxNx2 gradients for projX,Y
                dIdz.data<scalar_t>()         // BxNx1
            );                                // BxHxWx8
      }));
  output.push_back(dIdp);
  output.push_back(dIdz);

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("compute_visiblity_maps_cuda kernel failed: %s\n",
           hipGetErrorString(err));
    exit(-1);
  }
  return output;
}